#include "hip/hip_runtime.h"
// ray_intersect.cu
// Source: https://gamedev.stackexchange.com/a/103714/130059
__device__ float rayBoxIntersect ( float3 rpos, float3 rdir, float3 vmin, float3 vmax )
{
   float t[10];
   t[1] = (vmin.x - rpos.x)/rdir.x;
   t[2] = (vmax.x - rpos.x)/rdir.x;
   t[3] = (vmin.y - rpos.y)/rdir.y;
   t[4] = (vmax.y - rpos.y)/rdir.y;
   t[5] = (vmin.z - rpos.z)/rdir.z;
   t[6] = (vmax.z - rpos.z)/rdir.z;
   t[7] = fmax(fmax(fmin(t[1], t[2]), fmin(t[3], t[4])), fmin(t[5], t[6]));
   t[8] = fmin(fmin(fmax(t[1], t[2]), fmax(t[3], t[4])), fmax(t[5], t[6]));
   t[9] = (t[8] < 0 || t[7] > t[8]) ? NOHIT : t[7];
   return t[9];
}